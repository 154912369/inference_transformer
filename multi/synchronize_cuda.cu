#include "hip/hip_runtime.h"
#include "multi/synchronize_cuda.h"
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <stdint.h>
#include <sstream>
char* common_path = "./log/cuda_id.txt";
#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)
SynchronizeCUDA::SynchronizeCUDA(int myRank, int nRanks, int localRank){
   _myRank=myRank;
   _nRanks = nRanks;
   _localRank = localRank;
}
void SynchronizeCUDA::syn(){
    std::ofstream file_result(("./log/work."+std::to_string(_myRank)+".txt").c_str());
    if(file_result.is_open()) {
        
        file_result<<getpid();
        file_result.close();
    } 
    if(_myRank==0){
        ncclGetUniqueId(&_id);
        std::ofstream file(common_path);
        if(file.is_open()) {
           
            file << getpid();
            file<<"\n";
            file.write ((char *) _id.internal, sizeof(_id.internal) * sizeof(char));
            file.close();
        } else{
            printf("open failure as expected: %s\n\n\n\n\n\n",strerror(errno));
        
        }
        printf("main rank %d syncronize nccl id size is %d\n", _myRank, sizeof(_id.internal));
         sleep(1);
    }else{
        sleep(1);
        int ppid = getppid();
        std::string line;
        int pid;
        int try_size = 1;
        while(pid!=ppid&&try_size<10){
            std::ifstream file(common_path);
            if(file.is_open()) {
                getline(file, line);
                if(line.size()>0){
                    pid = std::stoi(line);
                    printf("get parent pid is %d ppid is %d\n",pid,ppid);
                    std::stringstream buffer;
                    buffer << file.rdbuf();
                    line = buffer.str();
                    sleep(1);
                    try_size +=1;
                }

            }
            file.close();
        }
        const char* result = line.c_str();
        for(int i=0;i<line.size();i++){
            _id.internal[i]=result[i];
        }
        printf("rank %d syncronize nccl id size is %d\n", _myRank, line.c_str());

    }
    //picking a GPU based on localRank, allocate device buffers
    CUDACHECK(hipSetDevice(_localRank));
      //initializing NCCL
    NCCLCHECK(ncclCommInitRank(&_comm, _nRanks, _id, _myRank));
    CUDACHECK(hipStreamCreate(&_stream));

}

ncclUniqueId& SynchronizeCUDA::getNcclId(){
    return _id;
}

int SynchronizeCUDA::get_rank(){
    return _myRank;
}


SynchronizeCUDA::~SynchronizeCUDA(){
      //finalizing NCCL
  ncclCommDestroy(_comm);
  printf("start to destroy rank %d\n",_myRank);
}

void SynchronizeCUDA::AllReduce(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op){

  //communicating using NCCL
  NCCLCHECK(ncclAllReduce(sendbuff, recvbuff, count,  datatype, op, _comm, _stream));
  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(_stream));

}

void SynchronizeCUDA::BroadCast(const void* sendbuff, void* recvbuff, size_t count, 
    ncclDataType_t datatype){

  //communicating using NCCL
  NCCLCHECK(ncclBroadcast(sendbuff, recvbuff, count,  datatype, 0, _comm, _stream));
  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(_stream));

}
